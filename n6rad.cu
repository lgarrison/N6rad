#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <chrono>
#include <assert.h>
using namespace std::chrono; 

// Problem parameters
// TODO: Set at runtime (check performance)
#define N 256
#define M 32
#define K (N/M)
#define T 64
#define M3 (M*M*M)
#define N3 ((int64_t)N*N*N)
#define K3 (K*K*K)

/*

TODO
- Multiple streams
- Pinned memory
- Real radiative transfer function
- Could consider pencil-on-block or pencil-on-pencil if CPU memory is getting out of hand.
*/

#define cudaCheckErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template<typename FLOAT>
__host__ __device__ inline FLOAT pair_op(FLOAT sink, FLOAT source, int dist2){
    // faster to multiply by (1/r^2) than divide by r^2
    return 2*((2*sink + 1) * (2*source + 1))*(1/static_cast<FLOAT>(dist2));
}

// Take 1D index i
// and convert it to the corresponding 3D (ix,iy,iz) index
// expand lets you multiply the final result by some factor (useful for converting block indices to cell indices)
__host__ __device__ inline void unravel_index(int i, int dim, int &ix, int &iy, int &iz, int expand=1){
    ix = i / (dim*dim);
    iy = i / dim - ix*dim;
    iz = i % dim;
    
    ix *= expand;
    iy *= expand;
    iz *= expand;
}

// Apply one source block to one sink block
// Cells must be in block order, i.e. (X,Y,Z,x,y,z)
template<typename FLOAT>
__global__ void block_on_block(FLOAT *cells, FLOAT *partial_sums, int source_pencil){
    int tid = threadIdx.x;
    int sinkblock = blockIdx.x;  // global 1D block index
    int sourcez = blockIdx.y;    // z offset of source block in source pencil
    int sourceblock = source_pencil*K + sourcez;  // global 1D
    
    // Get the (i,j,k) index of the sink and source block in the unpermuted N^3 grid
    int _sinki, _sinkj, _sinkk;
    int _sourcei, _sourcej, _sourcek;
    unravel_index(sinkblock, K, _sinki, _sinkj, _sinkk, M);
    unravel_index(sourceblock, K, _sourcei, _sourcej, _sourcek, M);
    
    FLOAT *sinks = cells + M3*sinkblock;
    FLOAT *sources = cells + M3*sourceblock;
    FLOAT *partials = partial_sums + N3*sourcez + M3*sinkblock;  // partials for this K,cell
    
    __shared__ FLOAT source_cache[T];
    // Sink loop
    for(int i = tid; i < M3; i += T){
        FLOAT thissink = sinks[i];
        FLOAT res = 0;
        
        // Get the location within the block
        int sinki, sinkj, sinkk;
        unravel_index(i, M, sinki, sinkj, sinkk);
        
        // and add on the coordinates of the block
        sinki += _sinki;
        sinkj += _sinkj;
        sinkk += _sinkk;
        
        // Source loop
        for(int j = 0; j < M3; j += T){
            // Each thread loads one source
            source_cache[tid] = sources[j+tid];
            __syncthreads();

            // Each thread loops over all sources
            for(int t = 0; t < T; t++){
                // TODO: there's a few options to optimize this indexing math
                // - get rid of mod
                // - if T divides M^2, only have to update j,k
                // - can precompute index(es) into shared array
                // - could use bit shifts, but probably don't want to constrain to power of 2
                
                // Get the location within the block
                int sourcei, sourcej, sourcek;
                unravel_index(j+t, M, sourcei, sourcej, sourcek);

                // and add on the coordinates of the block
                sourcei += _sourcei;
                sourcej += _sourcej;
                sourcek += _sourcek;
                
                int dist2 = (sinki - sourcei)*(sinki - sourcei) + (sinkj - sourcej)*(sinkj - sourcej) + (sinkk - sourcek)*(sinkk - sourcek);
                
                // A dummy function, just trying to force some floating point math
                res += pair_op(thissink, source_cache[t], dist2);
            }
            // We change the source cache at the top of the loop; sync here
            __syncthreads();
        }
        
        partials[i] = res;
        __syncthreads();
    }
}

using FLOAT = float;

void do_cpu(FLOAT *cells, FLOAT *result){
    // Do the same calculation on the GPU
    // We'll assume the cells are not permuted into blocks; i.e. standard C row-major order
    // This will provide some insurance against repeated indexing errors
    
    #pragma omp parallel for schedule(static)
    for(int i = 0; i < N3; i++){
        result[i] = 0;
        int ix, iy, iz;
        unravel_index(i, N, ix, iy, iz);
        for(int j = 0; j < N3; j++){
            int jx, jy, jz;
            unravel_index(j, N, jx, jy, jz);
            int dist2 = (ix - jx)*(ix - jx) + (iy - jy)*(iy - jy) + (iz - jz)*(iz - jz);
            result[i] += pair_op(cells[i], cells[j], dist2);
        }
    }
}

// The cells are laid out so that individual blocks are physically contiguous for the GPU
// Unpermute them into standard C order
void unpermute_cells(FLOAT *unpermuted_cells, FLOAT *cells){
    #pragma omp parallel for schedule(static)
    for(int i = 0; i < N; i++){
        int ki = i / M;
        int mi = i % M;
        
        for(int j = 0; j < N; j++){
            int kj = j / M;
            int mj = j % M;
            
            for(int k = 0; k < N; k++){
                int kk = k / M;
                int mk = k % M;
                
                int bstart = (ki*K*K + kj*K + kk)*M3;
                int off = mi*M*M + mj*M + mk;
                int pi = bstart + off;
        
                int to = i*N*N + j*N + k;
                
                unpermuted_cells[to] = cells[pi];
            }
        }
    }
}

void check_cpu(FLOAT *cells, FLOAT *gpu_result){
    FLOAT *cpu_result = new FLOAT[N3];
    
    // Make an unpermuted version for the CPU to operate on
    FLOAT *unpermuted_cells = new FLOAT[N3];
    unpermute_cells(unpermuted_cells, cells);
    
    auto start = high_resolution_clock::now(); 
    do_cpu(unpermuted_cells, cpu_result);
    auto elapsed = duration_cast<nanoseconds>(high_resolution_clock::now() - start);
    printf("CPU execution took %.3g seconds\n", elapsed.count()/1e9);
    
    // Reorder the GPU result into the order that the CPU produces
    FLOAT *unpermuted_gpu_result = new FLOAT[N3];
    unpermute_cells(unpermuted_gpu_result, gpu_result);
    
    for(int i = 0; i < N3; i++){
        FLOAT rerr = std::abs((cpu_result[i] - unpermuted_gpu_result[i])/cpu_result[i]);
        if(rerr > 1e-4){
            printf("Error! cpu_result[%d] = %g, gpu_result[%d] = %g, rerr = %g\n", i, cpu_result[i], i, unpermuted_gpu_result[i], rerr);
            break;
        }
        
        if(i == N3-1)
            printf("GPU result matches CPU result!\n");
    }
    
    delete[] unpermuted_cells;
    delete[] unpermuted_gpu_result;
    delete[] cpu_result;
}

// Host driver
int main(int argc, char **argv){
    // check params
    assert(M*K == N);
    assert((M3/T)*T == M3);
    assert((T/32)*32 == T);

    // Allocate and fill the cells
    FLOAT *cells = new FLOAT[N3];
    FLOAT *result = new FLOAT[N3];
    for(int64_t i = 0; i < N3; i++){
        cells[i] = ((double) rand()) / RAND_MAX;
        result[i] = 0;
    }
    
    FLOAT **partials = new FLOAT*[K*K];
    for(int i = 0; i < K*K; i++){
        partials[i] = new FLOAT[N3*K];
    }
    
    hipProfilerStart();
    auto start = high_resolution_clock::now();
    
    // the device-side arrays
    FLOAT *dev_cells, *dev_partial_sums;
    cudaCheckErrors(hipMalloc(&dev_cells, sizeof(FLOAT)*N3));
    // we're launching as pencil on cube, so we need a pencil's worth of partials for each cell
    cudaCheckErrors(hipMalloc(&dev_partial_sums, sizeof(FLOAT)*N3*K));
    
    // send over the cells
    cudaCheckErrors(hipMemcpy(dev_cells, cells, sizeof(FLOAT)*N3, hipMemcpyHostToDevice));
    
    for(int i = 0; i < K*K; i++){
        dim3 grid(K3,K);
        dim3 block(T);
        block_on_block<<<grid,block>>>(dev_cells, dev_partial_sums, i);
        
        // Get the result
        cudaCheckErrors(hipMemcpy(partials[i], dev_partial_sums, sizeof(FLOAT)*N3*K, hipMemcpyDeviceToHost));
        cudaCheckErrors(hipDeviceSynchronize());
    }
    
    auto elapsed = duration_cast<nanoseconds>(high_resolution_clock::now() - start);
    printf("GPU took %.3g seconds for N=%d, M=%d\n", elapsed.count()/1e9, N, M);
    
    hipProfilerStop();
    
    // Now sum the K^3 results per cell
    start = high_resolution_clock::now();
    #pragma omp parallel for schedule(static)
    for(int64_t i = 0; i < N3; i++){
        for(int64_t j = 0; j < K*K; j++){
            for(int64_t kk = 0; kk < K; kk++){
                result[i] += partials[j][kk*N3 + i];
            }
        }
    }
    elapsed = duration_cast<nanoseconds>(high_resolution_clock::now() - start);
    printf("Reduction took %.3g seconds\n", elapsed.count()/1e9);
    
    // Anything much bigger than this is unlikely to complete on the CPU in a useful amount of time
    if(N <= 64)
        check_cpu(cells, result);
    
    // Verify result, only useful if the cells are filled with a constant
    /*printf("result[0] = %g\n", result[0]);
    for(int i = 0; i < N3; i++){
        if (result[i] != 18.*N3){
            printf("result[%d] = %g\n", i, result[i]);
            break;
        }
        if(i == N3-1)
            printf("Verified!\n");
    }
    */
    
    cudaCheckErrors(hipFree(dev_cells));
    cudaCheckErrors(hipFree(dev_partial_sums));
    
    for(int i = 0; i < K*K; i++)
        delete[] partials[i];
    delete[] partials;
    
    delete[] cells;
    
    return 0;
}
